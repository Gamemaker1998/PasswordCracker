#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <cmath>
#include <Windows.h>

const int CUDA_MAX_BLOCKS = pow(2, 16);

struct configStruct
{
	hipDeviceProp_t properties;
	dim3 threads;
	dim3 blocks;
	unsigned long long totalThreads;
	unsigned long long threadsPerLoop;
	unsigned long long loops;
	unsigned long long totalBlocks;
	unsigned int totalRegistersPerFunction = 7;
	unsigned long long totalRegistersPerBlock;
}mainConfig;

void showSpecs();
void initRequirements(int, int);


/* Funzt aber Speicher ineffizient
__global__ void crack(char* d_password, char* d_passwordPerThread, int digits, int charset, long distributor, char* endPassword, bool* cracked)
{
	int x = threadIdx.x;
	int index = x * digits;

	if(x >= distributor)
		return;

	int number = x;

	for(int i=0;  i<digits && distributor > 0;  i++)
	{
		d_passwordPerThread[index + i] = (int)(number / (distributor / charset)) + 'A';
		distributor = (int)(distributor / charset);
		number = (int)(number % distributor);

		if(d_passwordPerThread[index + i] != d_password[i])
			return;
	}

	memcpy(endPassword, &(d_passwordPerThread[index]), sizeof(char) * digits);
	*cracked = true;
}*/


__global__ void test(unsigned long long* number, bool* guessed)
{
	unsigned long long id = (unsigned long long)((blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

	unsigned long long a = 1;
	unsigned long long b = -2;
	unsigned long long erg = a + b;

	if(id == *number)
		*guessed = true;
}

/*int main(int argc, char* argv[])
{
	dim3 blocks(65535);
	dim3 threads(1024, 1, 1);

	unsigned long long number = 0;
	bool guessed = false;

	scanf("%llu", &number);

	unsigned long long* d_number;
	bool* d_guessed;

	hipMalloc(&d_number, sizeof(long long));
	hipMemcpy(d_number, &number, sizeof(long long), hipMemcpyHostToDevice);

	hipMalloc(&d_guessed, sizeof(bool));

	test<<<blocks, threads>>>(d_number, d_guessed);
	hipError_t error = hipGetLastError();
	printf("\n%s: %s\n\n", hipGetErrorName(error), hipGetErrorString(error));

	hipMemcpy(&guessed, d_guessed, sizeof(bool), hipMemcpyDeviceToHost);

	if(guessed)
		printf("\nDie Zahl %llu wurde erraten\n\n", number);
	else
		printf("\nDie Zahl %llu wurde nicht erraten\n\n", number);

	system("PAUSE");
}*/


__global__ void crack(char* d_password, int digits, int charset, unsigned long long distributor, char* endPassword, bool* cracked, unsigned int factor)
{
	unsigned long long id = (unsigned long long)((blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);
	id *= factor;

	if (id >= distributor)
		return;

	// Werte sichern
	unsigned long long distributorSave = distributor;

	unsigned long long number;
	bool truePassword = false;
	char currentCharacter = '\0';

	for(int j=0;  j<factor;  j++, id++)
	{
		number = id;
		truePassword = true;
		distributor = distributorSave;
		for (int i = 0; i<digits && distributor > 0; i++)
		{
			currentCharacter = (char)(number / (distributor / charset)) + 'A';
			distributor = (unsigned long long)(distributor / charset);
			number = (unsigned long long)(number % distributor);

			if (currentCharacter != d_password[i])
			{
				truePassword = false;
				break;
			}
		}
		if(truePassword)
			break;
	}

	if(truePassword)
	{
		distributor = distributorSave;
		number = id;
		for (int i = 0; i<digits && distributor > 0; i++)
		{
			endPassword[i] = (int)(number / (distributor / charset)) + 'A';
			distributor = (int)(distributor / charset);
			number = (int)(number % distributor);
		}
		*cracked = true;
	}
}


__host__ bool init(char* password, int digits, int pwCharset, char* endPassword)
{
	char* d_password;
	char* d_endPassword;
	bool* d_cracked;

	bool h_cracked = false;

	initRequirements(digits, pwCharset);

	showSpecs();

	hipMalloc(&d_password, sizeof(char) * digits);
	hipMemcpy(d_password, password, digits * sizeof(char), hipMemcpyHostToDevice);

	hipMalloc(&d_endPassword, sizeof(char) * digits);

	hipMalloc(&d_cracked, sizeof(bool));

	printf("Password to crack: %.*s\n", digits, password);
	
	crack<<<mainConfig.blocks, mainConfig.threads>>>(d_password, digits, pwCharset, mainConfig.totalThreads, d_endPassword, d_cracked, mainConfig.loops);

	hipMemcpy(endPassword, d_endPassword, sizeof(char) * digits, hipMemcpyDeviceToHost);
	hipMemcpy(&h_cracked, d_cracked, sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(d_password);
	hipFree(d_endPassword);
	hipFree(d_cracked);

	return h_cracked;
}

void showSpecs()
{
	printf("\nGPU CUDA specs:\n\tName: %s\n\tShared mem per block: %uB\n\tTotal global mem: %uB\n", mainConfig.properties.name, mainConfig.properties.sharedMemPerBlock, mainConfig.properties.totalGlobalMem);
	printf("\tRegisters per block: %u\n", mainConfig.properties.regsPerBlock);
	printf("\n\n");
	printf("Requirements:\n");
	printf("\tNumber of blocks: %llu\n\tTotal length: %llu\n\tMaximum length: %u\n", mainConfig.totalBlocks, mainConfig.totalThreads, mainConfig.properties.totalGlobalMem);
	printf("\tBlock dimensioning\n\t\ty: %u\n\t\tx: %u\n", mainConfig.blocks.y, mainConfig.blocks.x);
	printf("\tRegisters per Block: %llu\n\tRegisters per function: %u\n", mainConfig.totalRegistersPerBlock, mainConfig.totalRegistersPerFunction);
	printf("\tLoops: %llu\n\tThreads per loop: %llu\n", mainConfig.loops, mainConfig.threadsPerLoop);
	printf("\n\n");
}

void initRequirements(int digits, int pwCharset)
{
	int deviceID;
	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&mainConfig.properties, deviceID);

	mainConfig.totalThreads = (unsigned long long)(pow(pwCharset, digits));

	mainConfig.totalBlocks = (unsigned long long)(1 + mainConfig.totalThreads / mainConfig.properties.maxThreadsPerBlock);
	mainConfig.threads = {(unsigned int)mainConfig.properties.maxThreadsPerBlock, (unsigned int)1, (unsigned int)1};

	unsigned int value = pow(2, 16) - 1;

	mainConfig.blocks.z = 1;
	mainConfig.blocks.y = 1;
	if(mainConfig.totalBlocks > value)
		mainConfig.blocks.x = value;
	else
		mainConfig.blocks.x = mainConfig.totalBlocks;

	mainConfig.loops = (unsigned long long)(1 + mainConfig.totalBlocks / value);
	mainConfig.threadsPerLoop = (unsigned long long)(mainConfig.blocks.x * mainConfig.threads.x);
	
	mainConfig.totalRegistersPerBlock = (mainConfig.threads.x * mainConfig.threads.y * mainConfig.threads.z) * mainConfig.totalRegistersPerFunction;
}

